#include "hip/hip_runtime.h"
#include  "init_cuda.h"
#define N 10
#include "book.h"

__global__ void add(int *a, int *b, int *c);

void InitCUDA()
{
    int a[N],b[N],c[N];
    //CPUBitmap bitmap(16,16);
    int *dev_a,*dev_b,*dev_c;
    HANDLE_ERROR(hipMalloc((void **)&dev_a, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, N*sizeof(int)));
    for(int i=0;i<N;i++)
    {
        a[i]=-i;
        b[i]=i*i;
    }
    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int),hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int),hipMemcpyHostToDevice));
    //HANDLE_ERROR(hipMemcpy(dev_c, c, N*sizeof(int),hipMemcpyHostToDevice));
    add<<<N, 1>>>(dev_a,dev_b,dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int),hipMemcpyDeviceToHost));
    for(int i=0;i<N;i++)
    {
        printf("%d + %d= %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
__global__ void add(int *a, int *b, int *c)
{
    int tid =blockIdx.x;
    if(tid<N)
        c[tid] = a[tid]+b[tid];

}
