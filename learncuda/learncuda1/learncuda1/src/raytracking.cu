#include "hip/hip_runtime.h"
#include  "raytracking.h"

#include "book.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define INF 2e10f
#define rnd(x) (x*rand()/RAND_MAX)
#define SPHEREs 20

struct Sphere
{
    float r,g,b;
    float radius;
    float x,y,z;
    __device__ float hit(float ox, float oy, float *n)
    {
        float dx = ox-x;
        float dy = oy-y;
        if(dx*dx + dy*dy < radius*radius)
        {
            float dz = sqrtf(radius*radius - dx*dx - dy*dy);
            *n = dz/sqrtf(radius*radius);
            return dz+z;
        }
        return -INF;
    }
};

__global__ void kenel(unsigned char *ptr);

__constant__ Sphere s[SPHEREs];
void ray()
{
    hipEvent_t start,stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start,0));

    CPUBitmap bitmap(DIM,DIM);
    unsigned char *dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&s,sizeof(Sphere)*SPHEREs));

    Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere)*SPHEREs);
    for(int i=0;i<SPHEREs;i++)
    {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(1000.0f) - 500;
        temp_s[i].y = rnd(1000.0f) - 500;
        temp_s[i].z = rnd(1000.0f) - 500;
        temp_s[i].radius = rnd(100.0f) + 20;
    }

    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere)*SPHEREs));
    free(temp_s);

    dim3 grid(DIM/16,DIM/16);
    dim3 threads(16,16);

    kenel<<<grid,threads>>>(dev_bitmap);
    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(),hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));
    printf("Time to generate: %3.1f ms\n", elapsedTime);

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    bitmap.display_and_exit();
    HANDLE_ERROR(hipFree(dev_bitmap));
    HANDLE_ERROR(hipFree(s));
}
__global__ void kenel(unsigned char *ptr)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int offset = x + y*blockDim.x*gridDim.x;

    float ox = (x-DIM/2);
    float oy = (y-DIM/2);

    float r=0,b=0,g=0;
    float maxz=-INF;
    for(int i=0;i<SPHEREs;i++)
    {
        float n;
        float t = s[i].hit(ox,oy,&n);
        if(t>maxz)
        {
            float fscale = n;
            r = s[i].r*fscale;
            g = s[i].g*fscale;
            b = s[i].b*fscale;
        }
    }


    ptr[offset*4+0] = (int)(r*255);
    ptr[offset*4+1] = (int)(g*255);
    ptr[offset*4+2] = (int)(b*255);
    ptr[offset*4+3] = 255;
}
